
#include <hip/hip_runtime.h>
/*
 * JCuda - Java bindings for NVIDIA CUDA driver and runtime API
 * http://www.jcuda.org
 *
 *
 * This code is based on the NVIDIA 'reduction' CUDA sample,
 * Copyright 1993-2010 NVIDIA Corporation.
 */
extern "C"
__global__ void sumfloat(float *g_idata,float *g_odata, unsigned int n)
{
	extern __shared__ float sdata[]; 
    unsigned int tid = threadIdx.x; // thread courant dans le block
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; // index g�n�ral
	sdata[tid] = g_idata[i]; // copy vers la shared memory du block
	__syncthreads(); // on attends tous les blocks
	
	if (i >= n) return; // on coupe au dela du cutoff
	// do reduction in shared mem for one block 
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) { // si correspond � un multiple de la dimension
		sdata[tid] += sdata[tid + s];
		}
		__syncthreads(); // on attends
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0]; 
}

extern "C"
__global__ void minfloat(float *g_idata,float *g_odata, unsigned int n)
{
	extern __shared__ float sdata[]; 
    unsigned int tid = threadIdx.x; // thread courant dans le block
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; // index g�n�ral
	sdata[tid] = g_idata[i]; // copy vers la shared memory du block
	__syncthreads(); // on attends tous les blocks
	
	if (i >= n) return; // on coupe au dela du cutoff
	// do reduction in shared mem for one block 
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) { // si correspond � un multiple de la dimension
			if (sdata[tid+s]<sdata[tid]) {
				sdata[tid] = sdata[tid + s];
			}
		}
		__syncthreads(); // on attends
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0]; 
}

extern "C"
__global__ void maxfloat(float *g_idata,float *g_odata, unsigned int n)
{
	extern __shared__ float sdata[]; 
    unsigned int tid = threadIdx.x; // thread courant dans le block
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; // index g�n�ral
	sdata[tid] = g_idata[i]; // copy vers la shared memory du block
	__syncthreads(); // on attends tous les blocks
	
	if (i >= n) return; // on coupe au dela du cutoff
	// do reduction in shared mem for one block 
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) { // si correspond � un multiple de la dimension
			if (sdata[tid+s]>sdata[tid]) {
				sdata[tid] = sdata[tid + s];
			}
		}
		__syncthreads(); // on attends
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0]; 
}
extern "C" 
__global__ void stringfilter(char2 *text,unsigned int char_per_word,unsigned int n) 
{
	unsigned int id = threadIdx.x; // numero de thread courant
	unsigned int index = blockIdx.x * blockDim.x + id; // index absolu
	
	//recherche du texte
	unsigned int offset = index * char_per_word;
	if (index <n) {
		printf(" texte %c %d",text[offset],index);
	}
	
}